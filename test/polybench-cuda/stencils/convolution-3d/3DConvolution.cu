#include "hip/hip_runtime.h"
/**
 * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */
#include "3DConvolution.cuh"
#include "3DConvolution_sh_ann.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

extern "C" __global__ void convolution3D_kernel(int ni, int nj, int nk, ANN_A DATA_TYPE* A, ANN_B DATA_TYPE* B, int i)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;


	if ((i < (_PB_NI-1)) && (j < (_PB_NJ-1)) &&  (k < (_PB_NK-1)) && (i > 0) && (j > 0) && (k > 0))
	{
		B[i*(NK * NJ) + j*NK + k] = c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c21 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c23 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c31 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c33 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c12 * A[(i + 0)*(NK * NJ) + (j - 1)*NK + (k + 0)]  +  c22 * A[(i + 0)*(NK * NJ) + (j + 0)*NK + (k + 0)]   
					     +   c32 * A[(i + 0)*(NK * NJ) + (j + 1)*NK + (k + 0)]  +  c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  
					     +   c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  +  c21 * A[(i - 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  
					     +   c23 * A[(i + 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  +  c31 * A[(i - 1)*(NK * NJ) + (j + 1)*NK + (k + 1)]  
					     +   c33 * A[(i + 1)*(NK * NJ) + (j + 1)*NK + (k + 1)];
	}
}
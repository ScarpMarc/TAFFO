#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */
#include "jacobi2D.cuh"
#include "jacobi2D_sh_ann.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

extern "C" __global__ void runJacobiCUDA_kernel1(int n, ANN_A DATA_TYPE* A, ANN_B DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		B[i*N + j] = 0.2f * (A[i*N + j] + A[i*N + (j-1)] + A[i*N + (1 + j)] + A[(1 + i)*N + j] + A[(i-1)*N + j]);	
	}
}


extern "C" __global__ void runJacobiCUDA_kernel2(int n, ANN_A DATA_TYPE* A, ANN_B DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		A[i*N + j] = B[i*N + j];
	}
}

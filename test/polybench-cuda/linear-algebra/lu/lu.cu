#include "hip/hip_runtime.h"
/**
 * lu.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */
#include "lu.cuh"
#include "lu_sh_ann.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>


extern "C" __global__ void lu_kernel1(int n, ANN_A DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((j > k) && (j < _PB_N))
	{
		A[k*N + j] = A[k*N + j] / A[k*N + k];
	}
}


extern "C" __global__ void lu_kernel2(int n, ANN_A DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i > k) && (j > k) && (i < _PB_N) && (j < _PB_N))
	{
		A[i*N + j] = A[i*N + j] - A[i*N + k] * A[k*N + j];
	}
}

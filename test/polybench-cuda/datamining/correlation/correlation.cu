#include "hip/hip_runtime.h"
/**
 * correlation.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */
#include "correlation.cuh"
#include "correlation_sh_ann.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>


extern "C" __global__ void mean_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *data ANN_DATA,  DATA_TYPE float_n ANN_FLOAT_N)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < m)
	{
    	DATA_TYPE __attribute__((annotate("scalar(range(0, 3000))"))) accum = 0.0;
		accum = 0.0;

		int i;
		for (i=0; i < n; i++)
		{
			accum += data[i*m + j];
		}
		
		mean[j] = accum / float_n;
	}
}


extern "C" __global__ void std_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *std ANN_STD, DATA_TYPE *data ANN_DATA, DATA_TYPE float_n ANN_FLOAT_N, DATA_TYPE eps ANN_EPS)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < m)
	{
    DATA_TYPE __attribute__((annotate("scalar(range(0, 3000))"))) accum = 0.0;
		
		int i;
		for (i = 0; i < n; i++)
		{
      accum += (data[i*m + j] - mean[j]) * (data[i*m + j] - mean[j]);
		}
    std[j] = sqrt(accum / float_n);
		if(std[j] <= eps) 
		{
			std[j] = 1.0;
		}
	}
}


extern "C" __global__ void reduce_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *std ANN_STD, DATA_TYPE *data ANN_DATA, DATA_TYPE float_n ANN_FLOAT_N)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < n) && (j < m))
	{
		data[i*m + j] -= mean[j];
    data[i*m + j] /= sqrt(float_n) * std[j];
	}
}	


extern "C" __global__ void corr_kernel(int m, int n, DATA_TYPE *symmat ANN_SYMMAT, DATA_TYPE *data ANN_DATA)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x;

	int i, j2;
	if (j1 < (_PB_M-1))
	{
		symmat[j1*M + j1] = 1.0;

		for (j2 = (j1 + 1); j2 < _PB_M; j2++)
		{
			//symmat[j1*M + j2] = 0.0;

			for(i = 0; i < _PB_N; i++)
			{
				symmat[j1*M + j2] += data[i*M + j1] * data[i*M + j2];
			}
			symmat[j2*M + j1] = symmat[j1*M + j2];
		}
	}
}
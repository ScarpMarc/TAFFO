#include "hip/hip_runtime.h"
/**
 * correlation.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */
#include "correlation.cuh"
#include "correlation_sh_ann.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

#define FLOAT_N 3214212.01f

#define EPS 0.005f

extern "C" __global__ void mean_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *data ANN_DATA)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < _PB_M)
	{
		mean[j] = 0.0;

		int i;
		for(i=0; i < _PB_N; i++)
		{
			mean[j] += data[i*M + j];
		}
	
		mean[j] /= (DATA_TYPE)FLOAT_N;
	}
}


extern "C" __global__ void std_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *std ANN_STD, DATA_TYPE *data ANN_DATA)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < _PB_M)
	{
		std[j] = 0.0;

		int i;
		for(i = 0; i < _PB_N; i++)
		{
			DATA_TYPE __attribute__((annotate("scalar()"))) tmp = (data[i*m + j] - mean[j]);
			tmp = tmp * tmp;
			std[j] += tmp;
		}
		DATA_TYPE __attribute__((annotate("scalar(range(0,5000) final)"))) tmp = std[j];
		std[j] = sqrt(tmp / FLOAT_N);
		if(std[j] <= EPS) 
		{
			std[j] = 1.0;
		}
	}
}


extern "C" __global__ void reduce_kernel(int m, int n, DATA_TYPE *mean ANN_MEAN, DATA_TYPE *std ANN_STD, DATA_TYPE *data ANN_DATA)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < _PB_N) && (j < _PB_M))
	{
		data[i*M + j] -= mean[j];
		__attribute__((annotate("scalar(range(-100, 100) final)"))) DATA_TYPE tmp = sqrt(FLOAT_N) * std[j];
		data[i*m + j] /= tmp;
	}
}


extern "C" __global__ void corr_kernel(int m, int n, DATA_TYPE *symmat ANN_SYMMAT, DATA_TYPE *data ANN_DATA)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x;

	int i, j2;
	if (j1 < (_PB_M-1))
	{
		symmat[j1*M + j1] = 1.0;

		for (j2 = (j1 + 1); j2 < _PB_M; j2++)
		{
			symmat[j1*M + j2] = 0.0;

			for(i = 0; i < _PB_N; i++)
			{
				symmat[j1*M + j2] += data[i*M + j1] * data[i*M + j2];
			}
			symmat[j2*M + j1] = symmat[j1*M + j2];
		}
	}
}